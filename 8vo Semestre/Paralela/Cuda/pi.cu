
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>

#define T_FLAG "-t"
#define BLOCK_SIZE 1024

int get_flag_value(int argc, char** argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

__global__ void piKernel(double* result) {
    int position = threadIdx.x + BLOCK_SIZE * blockIdx.x;
    double const symbol = -1;

    printf("f(rand_n): %d\n", position);

    result[position] = pow(symbol, position) * (4 / (1 + (2 * (double)position)));
}

void calculate_pi(int n_terms) {
    int t_blocks = (int)(ceil(n_terms / BLOCK_SIZE));
    int ceil_terms = t_blocks * BLOCK_SIZE;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(t_blocks);

    size_t result_size = sizeof(double) * ceil_terms;
    double* result = (double*)malloc(result_size);
    double* d_result;

    hipMalloc(&d_result, result_size);
    piKernel<<<blocksPerGrid, threadsPerBlock>>>(d_result);

    hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    double pi = 0;
    for (int i = 0; i < ceil_terms; i++) {
        pi += result[i];
    }

    printf("pi: %.20f\n", pi);
}

int main(int argc, char** argsv) {
    calculate_pi(get_flag_value(argc, argsv, T_FLAG));
}
