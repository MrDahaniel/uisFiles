
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>

// Definimos las constantes globales de CUDA

#define N_FLAG "-n"
#define BLOCK_SIZE 32

int get_flag_value(int argc, char** argsv, std::string flag) {
    bool arg = false;
    int n = -1;

    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0 && !arg) {
            n = std::stoi(argsv[i + 1]);
            arg = true;
        } else {
            continue;
        }
    }

    return n;
}

// __global__ void test

__global__ void testIfPrimeKernel(int number, bool** result_matrix) {
    // int div = 1;
    // int i = 1;
    bool is_prime = true;
    int i = threadIdx.y + 1;
    int j = threadIdx.x + 1;

    if (j < i && i % j == 0) {
        is_prime = false;
    }

    printf("darn");

    result_matrix[threadIdx.x][threadIdx.y] = is_prime;
}

void test_numbers(int number) {
    int new_number = ceil(number % BLOCK_SIZE) * BLOCK_SIZE;
    int grid_size = ceil(number / BLOCK_SIZE);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size, grid_size);

    size_t result_matrix_size = sizeof(bool) * pow(number, 2);
    bool** d_result_matrix;

    hipMalloc(&d_result_matrix, result_matrix_size);

    testIfPrimeKernel<<<blocksPerGrid, threadsPerBlock>>>(new_number, d_result_matrix);

    bool** result_matrix = (bool**)malloc(result_matrix_size);
    hipMemcpy(result_matrix, d_result_matrix, result_matrix_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < new_number; i++) {
        bool is_prime = true;

        for (int j = 0; j <= i; j++) {
            if (!result_matrix[j][i]) {
                is_prime = false;
                break;
            }
        }

        if (is_prime) {
            printf("%d is prime", i + 1);
        }
    }
}

int main(int argc, char** argsv) {
    int max_test_num = get_flag_value(argc, argsv, N_FLAG);

    test_numbers(max_test_num);
}
