
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>

// Definimos las constantes globales de CUDA

#define N_FLAG "-n"
#define BLOCK_SIZE 32

int get_flag_value(int argc, char** argsv, std::string flag) {
    bool arg = false;
    int n = -1;

    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0 && !arg) {
            n = std::stoi(argsv[i + 1]);
            arg = true;
        } else {
            continue;
        }
    }

    return n;
}

// __global__ void test

__global__ void testIfPrimeKernel(int number, bool* result_matrix) {
    int x_idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y_idx = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    int i = x_idx + 2;
    int j = y_idx + 2;

    result_matrix[x_idx * number + y_idx] = true;

    if (i > j && i % j == 0) {
        result_matrix[i * number + j] = false;
    } else {
        return;
    }
}

void test_numbers(int number) {
    int new_number = (double)(ceil((double)number / BLOCK_SIZE)) * 32;
    int grid_size = (double)(ceil((double)number / BLOCK_SIZE));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size, grid_size);

    size_t result_matrix_size = sizeof(bool) * pow(new_number, 2);
    bool* d_result_matrix;

    hipMalloc(&d_result_matrix, result_matrix_size);

    testIfPrimeKernel<<<blocksPerGrid, threadsPerBlock>>>(new_number, d_result_matrix);

    bool* result_matrix = (bool*)malloc(result_matrix_size);

    hipMemcpy(result_matrix, d_result_matrix, result_matrix_size, hipMemcpyDeviceToHost);

    for (int i = 2; i <= new_number; i++) {
        bool is_prime = true;

        for (int j = 2; j <= i; j++) {
            if (!result_matrix[i * new_number + j]) {
                is_prime = false;
                break;
            }
        }

        if (is_prime) {
            printf("%d is prime\n", i);
        }
    }
}

int main(int argc, char** argsv) {
    int max_test_num = get_flag_value(argc, argsv, N_FLAG);

    test_numbers(max_test_num);
}
