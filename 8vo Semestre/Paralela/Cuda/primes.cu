
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <cmath>
#include <string>
#include <tuple>

// Definimos las constantes globales de CUDA

#define N_FLAG "-n"
#define BLOCK_SIZE 32

int get_flag_value(int argc, char** argsv, std::string flag) {
    bool arg = false;
    int n = -1;

    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0 && !arg) {
            n = std::stoi(argsv[i + 1]);
            arg = true;
        } else {
            continue;
        }
    }

    return n;
}

__global__ void testIfPrimeKernel(int number, bool* result_matrix) {
    int x_idx, y_idx;
    int position = threadIdx.x + blockIdx.x * BLOCK_SIZE + 1;

    float p = (sqrtf(1 + 8 * position) - 1) / 2;
    double i0 = floor(p);

    if (i0 == p) {
        x_idx = i0;
        y_idx = i0;
    } else {
        x_idx = i0 + 1;
        y_idx = position - i0 * (i0 + 1) / 2;
    }

    x_idx -= 1;
    y_idx -= 1;

    // int kp = (BLOCK_SIZE * (BLOCK_SIZE + 1) / 2) - position;
    // int p = floor((sqrtf(1 + 8 * kp) - 1) / 2);
    // x_idx = BLOCK_SIZE - (kp - p * (p + 1) / 2);
    // y_idx = x_idx + 1 - (BLOCK_SIZE - p);

    printf("%d - (%d, %d)\n", position, x_idx, y_idx);

    result_matrix[x_idx * BLOCK_SIZE + y_idx] = true;

    if (x_idx > y_idx && y_idx != 1 && y_idx != 0 && x_idx % y_idx == 0) {
        result_matrix[x_idx * number + y_idx] = false;
    }
}

void test_numbers(int number) {
    int new_number = (int)(ceil((double)number / BLOCK_SIZE)) * BLOCK_SIZE;
    int tringular_size = (new_number * (new_number + 1) / 2);
    int grid_size = (int)(ceil((double)tringular_size / BLOCK_SIZE));

    printf("num: %d, t size: %d, grid: %d\n", new_number, tringular_size, grid_size);

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size);

    size_t result_matrix_size = sizeof(bool) * (tringular_size);
    bool* d_result_matrix;

    hipMalloc(&d_result_matrix, result_matrix_size);

    testIfPrimeKernel<<<blocksPerGrid, threadsPerBlock>>>(new_number, d_result_matrix);

    bool* result_matrix = (bool*)malloc(result_matrix_size);

    hipMemcpy(result_matrix, d_result_matrix, result_matrix_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < new_number; i++) {
        for (int j = 0; j < i; j++) {
            printf("%d ", result_matrix[i * new_number + j]);
        }
        printf(" - %d \n", i);
    }

    for (int i = 0; i <= number; i++) {
        bool is_prime = true;

        for (int j = 0; j <= i; j++) {
            if (!result_matrix[i * new_number + j]) {
                is_prime = false;
                break;
            }
        }

        if (is_prime) {
            printf("%d is prime\n", i);
        }
    }
}

int main(int argc, char** argsv) {
    int max_test_num = get_flag_value(argc, argsv, N_FLAG);

    test_numbers(max_test_num);
}
