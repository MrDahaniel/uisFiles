
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <string>

// Definimos las constantes globales de CUDA

#define N_FLAG "-n"

int get_flag_value(int argc, char** argsv, char* flag) {
    int n;

    for (int i = 0; i < argc; i++) {
        if (strcmp(argsv[i], flag) == 0) {
            n = atoi(argsv[i + 1]);
            break;
        }
    }

    return n;
}

// __global__ void test

__global__ void testIfPrimeKernel(int number, bool** result_matrix) {
    // int div = 1;
    // int i = 1;
    bool is_prime = true;
    int i = threadIdx.y;
    int j = threadIdx.x;

    if (j < i && i % j == 0) {
        is_prime = false;
    }

    result_matrix[threadIdx.x][threadIdx.y] = is_prime;
}

bool** test_numbers(int number) {
    bool** result_matrix = (bool**)malloc(sizeof(bool) * pow(number, 2));

    for (int i = 1; i <= 32; i++) {
        
    }

    dim3 threadsPerBlock();
    dim3 blocksPerGrid(number, number);

}

int main(int argc, char** argsv) {
    int max_test_num = get_flag_value(argc, argsv, N_FLAG);
}

