#include "hip/hip_runtime.h"
#include <string>

#include "aux.cuh"
#include "stdio.h"
#include "stdlib.h"

using namespace std;

// Definimos las constantes globales
#define G 6.67e-11
#define DELTA_T 86400
#define N_STEPS 3649
#define TOTAL_BODIES 2

// Definimos las constantes globales de CUDA
#define BLOCK_SIZE 1

typedef struct SpaceBody {
    int name;
    double3 acceleration;
    double3 speed;
    double3 position;
    double mass;

    void print() {
        printf("%d\n", name);
        printf("    Acceleration: %e, %e, %e\n",
               acceleration.x,
               acceleration.y,
               acceleration.z);
        printf("    Speed: %e, %e, %e\n",
               speed.x,
               speed.y,
               speed.z);
        printf("    Position: %e, %e, %e\n",
               position.x,
               position.y,
               position.z);
        printf("    Mass: %e\n", mass);
    }
} SpaceBody;

// Is this the kernel? Not sure...

__global__ void bodyNextStepKernel(SpaceBody* space_bodies, int num_bodies) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = 0; i < num_bodies; i++) {
        if (space_bodies[id].name == space_bodies[i].name) {
            continue;
        }

        double3 acc = (double3){0, 0, 0};

        double f =
            space_bodies[i].mass / pow(aux::magnitude_between_points(space_bodies[id].position, space_bodies[i].position), 2);
        double3 unit_vec = aux::get_unitary_vector(space_bodies[id].position, space_bodies[i].position);

        acc.x += -G * f * unit_vec.x;
        acc.y += -G * f * unit_vec.y;
        acc.z += -G * f * unit_vec.z;

        space_bodies[id].acceleration = acc;
    }

    space_bodies[id].speed.x += space_bodies[id].acceleration.x * DELTA_T;
    space_bodies[id].speed.y += space_bodies[id].acceleration.y * DELTA_T;
    space_bodies[id].speed.z += space_bodies[id].acceleration.z * DELTA_T;

    space_bodies[id].position.x += space_bodies[id].speed.x * DELTA_T;
    space_bodies[id].position.y += space_bodies[id].speed.y * DELTA_T;
    space_bodies[id].position.z += space_bodies[id].speed.z * DELTA_T;
}

SpaceBody* init_bodies() {
    SpaceBody* bodies = new SpaceBody[TOTAL_BODIES];

    bodies[0] = SpaceBody{
        0,
        double3{0, 0, 0},
        double3{0, 0, 0},
        double3{0, 0, 0},
        1.989e30,
    };

    bodies[1] = SpaceBody{
        1,
        double3{0, 0, 0},
        double3{0, 47.36e3, 0},
        double3{57.909e9, 0, 0},
        3.3011e23,
    };

    return bodies;
}

SpaceBody* simulate(SpaceBody* space_bodies) {
    // Sacamos la cantidad de cuerpos
    // size_t space_body_size = sizeof(SpaceBody);
    size_t space_bodies_size = sizeof(SpaceBody) * TOTAL_BODIES;

    // Definiendo cosas de cuda, esto no es lo más eficiente pero sólo
    // tenemos 5 cuerpos y eso no es divisible bonito
    dim3 threadsPerBlock(TOTAL_BODIES);
    dim3 numBlocks(1);

    // cargamos los cuerpos en el device, ¿creo?
    SpaceBody* d_space_bodies;
    hipMalloc(&d_space_bodies, space_bodies_size);
    hipMemcpy(d_space_bodies, space_bodies, space_bodies_size, hipMemcpyHostToDevice);

    for (int step = 0; step <= N_STEPS; step++) {
        bodyNextStepKernel<<<numBlocks, threadsPerBlock>>>(d_space_bodies, TOTAL_BODIES);
    }

    // Recuperamos los valores calculados
    SpaceBody* new_bodies = new SpaceBody[TOTAL_BODIES];

    hipMemcpy(new_bodies, d_space_bodies, space_bodies_size, hipMemcpyDeviceToHost);

    return new_bodies;
}

int main(int argc, char const* argv[]) {
    SpaceBody* space_bodies = init_bodies();
    SpaceBody* new_body = simulate(space_bodies);

    for (int i = 0; i < TOTAL_BODIES; i++) {
        new_body[i].print();
    }

    return 0;
}
