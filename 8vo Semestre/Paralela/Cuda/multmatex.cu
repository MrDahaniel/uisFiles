
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"
#include "math.h"
#include "time.h"

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

// thread block size
// #define BLOCK_SIZE 16
#define BLOCK_SIZE 4

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C);

// Matrix multiplication - host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// load A and B to devic memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	// allocate C in device memory 
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	printf("\n\n  dimensiones de C (antes del kernel): %d, %d\n",C.width,C.height);
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
	// invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C);
	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	// porqueria
	int i,j;
	printf("\n\n  dimensiones de C: %d, %d\n",C.height,C.width);
	for (i=0; i< C.width ; ++i) 
	for (j=0; j < C.height; ++j) {			
		//printf("\n\n%f", C.elements[i*C.width+j]); 
	}
	
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
	// each thread compute one element of C
	// by acumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e) {
		Cvalue = Cvalue + A.elements[row *A.width + e] * B.elements[e * B.width + col];
	}
	C.elements[row * C.width + col] = Cvalue;
}



int main( int argc, char** argv) {
	Matrix A, B, C;
	float MAT[8][8];
	float MAT2[8][8];
	float MATA[8][8];
	float MATB[8][8];
	// vamos al ejemplo de la matriz cuadrada, 8x8
	A.width = 8;
	B.width = 8;
	C.width = 8;
	A.height = 8;
	B.height = 8;
	C.height = 8;
	int count = 0;
	int i, j, k;
	float temp;
	printf("\n\n\n\n\n");
	printf("\n************************************\n");
	printf("Prueba de multiplicacion de matrices");
	printf("\n************************************\n");
	for (i=0; i< A.width ; ++i) {
		for (j=0; j < A.height; ++j) {
			temp = rand()%10*1.0;
			MAT[i][j]=temp;
			MATA[i][j]=temp;
		}
	}
	printf("\n\n -- Matriz A --\n");
	for (i=0; i< A.width ; ++i) {
		printf("\n");
		for (j=0; j < A.height; ++j) {
			printf(" %3.2f ",MAT[i][j]);
			count++;
		}
		printf("\n");
	}
	printf("\nContador = %d", count);
	size_t size = A.width*A.height*sizeof(float);
	A.elements = (float*)malloc(size);
	for (i=0; i< A.width ; ++i) {
		for (j=0; j < A.height; ++j) {
			A.elements[i*A.width+j]=MAT[i][j];
		}
	}
	// ahora la B
	count = 0;
	for (i=0; i< B.width ; ++i) {
		for (j=0; j < B.height; ++j) {
			temp = rand()%10*1.0;
			MAT[i][j]=temp;
			MATB[i][j]=temp ;
		}
	}
	printf("\n\n -- Matriz B --\n");
	for (i=0; i< B.width ; ++i) {
		printf("\n");
		for (j=0; j < B.height; ++j) {
			printf(" %3.2f ",MAT[i][j]);
			count++;
		}
		printf("\n");
	}
	printf("\nContador = %d", count);
	size = B.width*B.height*sizeof(float);
	B.elements = (float*)malloc(size);
	for (i=0; i< B.width ; ++i) {
		for (j=0; j < B.height; ++j) {
			B.elements[i*B.width+j]=MAT[i][j];
		}
	}
	// pasar la mat a la funcion MulMat
	C.elements = (float*)malloc(size);
	// aca hay que incluir el codigo que lleva control del tiempo
	clock_t tinicio, t_GPU;
	float tg,tc;
	tinicio=clock();
	//
	MatMul(A,B,C);
	//
	t_GPU=clock();
	tg = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\n\ntiempo de procesamiento (GPU): %6.3f s\n\n",tg);
	// aca se calculó el tiempo de la GPU, gracias Monica!
	for (i=0; i< C.width ; ++i) {
		for (j=0; j < C.height; ++j) {
			MAT[i][j]=C.elements[i*C.width+j];
		}
	}
	printf("\n -- Matrix resultante (GPU) --\n");
	for (i=0; i< C.width ; ++i) {
		printf("\n");
		for (j=0; j < C.height; ++j) {
			printf(" %3.2f ",MAT[i][j]);
		}
		printf("\n");
	}
	// aca vamos a realizar la multiplicacion de matrices mediante la cpu.
	// se analizaran los resultados.
	tinicio=clock();
	//
	for (i=0; i<8; ++i) {
		for (j=0; j<8; ++j) {
			MAT2[i][j]=0;
			for(k=0;k<8;k++) {
				MAT2[i][j]=MAT2[i][j]+MATA[i][k]*MATB[k][j];
			}
		}
	}
	//
	t_GPU=clock();
	tc = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\n\ntiempo de procesamiento (CPU): %6.3f s\n\n",tc);
	printf("\n -- Matrix resultante (CPU) --\n");
	for (i=0; i< C.width ; ++i) {
		printf("\n");
		for (j=0; j < C.height; ++j) {
			printf(" %3.2f ",MAT2[i][j]);
		}
		printf("\n");
	}
}
