
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>

#define SAMPLES_FLAG "-s"
#define LOWER_BOUND_FLAG "-l"
#define UPPER_BOUND_FLAG "-u"

#define BLOCK_SIZE 256
#define THREAD_LIMIT 43008
#define BLOCKS_ON_THREAD_LIMIT THREAD_LIMIT / BLOCK_SIZE

int get_flag_value(int argc, char** argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

__device__ float frand(int lower, int upper, int index, hiprandState* states) {
    return hiprand_uniform(&(states[index])) * (upper - lower) + lower;
}

__global__ void setupKernel(hiprandState* state, int seed) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, position, 0, &state[position]);
}

template <class Function>
__global__ void monteCarloKernel(
    int n_samples, int lower, int upper, Function fn,
    float* d_result, hiprandState* states) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    d_result[position] = fn(frand(lower, upper, position, states));
}

template <class Function>
float monte_carlo(int samples, int lower, int upper, Function fn) {
    /*
        Lo primero a realizar está en redondear la cantidad de números
        de la muestra a un número divisible por el tamaño de los bloques,
        a partir de esto también calcularemos loops. Esto de loops se hace
        con el fin de evadir una limitación respecto a la máxima cantidad
        de threads que podían ser usadas en cada ejecución.

        Básicamente, en el caso de superar las 43008 threads, la ejecución
        del programa daría como resultado 0. No estoy muy seguro de por qué
        pero lo hacía de manera local y en el colab. Entonces, para poder
        realizar la aproximación con más de 43008 threads, tengo que realiar
        el calculo por partes la aproximación. Es decir, de 43008 valores en
        43008 valores hasta alcanzar la sample size deseada. Esto implica un
        overhead terrible debido a la necesidad de lanzar múltiples veces el
        kernel pero es la única manera que encontré de "solucionar" este
        problema. Jaj
    */
    int t_blocks = (int)(ceil((float)samples / BLOCK_SIZE));
    int ceil_terms = t_blocks * BLOCK_SIZE;
    int loops = 1;

    if (ceil_terms > THREAD_LIMIT) {
        loops = (int)(ceil((float)ceil_terms / THREAD_LIMIT));
        t_blocks = BLOCKS_ON_THREAD_LIMIT;
        ceil_terms = THREAD_LIMIT;
    }

    float* aproxMonte = (float*)malloc(sizeof(float) * loops);

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(t_blocks);

    /*
        Ya dentro de este for es donde realizo el cálculo de los números
        al igual que al aproximación.
    */

    for (int loop = 0; loop < loops; loop++) {
        hiprandState* d_state;
        hipMalloc(&d_state, sizeof(hiprandState));

        size_t result_size = sizeof(float) * ceil_terms;

        float* d_result;
        hipMallocManaged(&d_result, result_size);

        /*
            Se definen general los números aleatorios a usar para la
            aproximación.
        */
        setupKernel<<<blocksPerGrid, threadsPerBlock>>>(d_state, rand());

        /*
            Y se evaluan y guardan en este kernel
        */
        monteCarloKernel<<<blocksPerGrid, threadsPerBlock>>>(
            samples, lower, upper, fn, d_result, d_state);

        hipDeviceSynchronize();

        /*
            se realiza la aproximación de monte carlo
            para esa iteración del loop
        */
        float monteSum = 0;
        for (int i = 0; i < ceil_terms; i++) {
            monteSum += d_result[i];
        }

        hipFree(d_state);
        hipFree(d_result);

        aproxMonte[loop] = monteSum / ceil_terms;
    }

    /*
        Y finalmente suman todas los valores de las iteraciones y
        se divide en la cantidad de loops.
    */
    float monteCarlo = 0.0;
    for (int i = 0; i < loops; i++) {
        monteCarlo += aproxMonte[i] / loops;
    }

    double total_terms = loops * ceil_terms;
    printf(
        "loops: %d, blocksPerGrid: %d, threadsPerBlock: %d, totalTerms: %.0f\n",
        loops, t_blocks, BLOCK_SIZE, total_terms);
    return monteCarlo;
}

int main(int argc, char** argsv) {
    srand(time(NULL));

    int n_samples = (int)get_flag_value(argc, argsv, SAMPLES_FLAG);
    int lower_bound = (int)get_flag_value(argc, argsv, LOWER_BOUND_FLAG);
    int upper_bound = (int)get_flag_value(argc, argsv, UPPER_BOUND_FLAG);

    auto fn = [] __device__(float x) { return powf(x, 2); };

    float monteCarlo = monte_carlo(n_samples, lower_bound, upper_bound, fn);

    float theoric = (powf(upper_bound, 3) / 3) - (powf(lower_bound, 3) / 3);
    float error = fabsf(theoric - monteCarlo) / theoric;

    printf("Monte Carlo: %.20f\nError: %.20f\n", monteCarlo, error);
}
