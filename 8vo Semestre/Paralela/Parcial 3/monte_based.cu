
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <string>

#define SAMPLES_FLAG "-s"
#define LOWER_BOUND_FLAG "-l"
#define UPPER_BOUND_FLAG "-u"

#define BLOCK_SIZE 64

int get_flag_value(int argc, char** argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

__device__ float frand(int lower, int upper, int index, hiprandState* states) {
    return hiprand_uniform(&(states[index])) * (upper - lower) + lower;
}

__global__ void setupKernel(hiprandState* state) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, position, 0, &state[position]);
}

__global__ void monteCarloKernel(int n_samples, int lower, int upper, float* d_result, hiprandState* states) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;

    // float n_rand = frand(lower, upper, position, states);
    // float n_rand = curand_uniform(&(states[position])) * (upper - lower) + lower;
    float n_rand = hiprand_uniform(&(states[position]));

    d_result[position] = powf(n_rand, 2);

    printf("Postion: %d, rand_n: %f, f(rand_n): %f\n", position, n_rand, d_result[position]);
}

void monte_carlo(int samples, int lower, int upper) {
    int t_blocks = (int)(ceil((float)samples / BLOCK_SIZE));
    int ceil_terms = t_blocks * BLOCK_SIZE;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(t_blocks);

    hiprandState* d_state;
    hipMalloc(&d_state, sizeof(hiprandState) * 2);

    size_t result_size = sizeof(float) * ceil_terms * 2;
    float* result = (float*)malloc(result_size);
    float* d_result;

    hipMalloc(&d_result, result_size);

    setupKernel<<<blocksPerGrid, threadsPerBlock>>>(d_state);
    monteCarloKernel<<<blocksPerGrid, threadsPerBlock>>>(samples, lower, upper, d_result, d_state);

    hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    float aproxMonte = 0;
    for (int i = 0; i < ceil_terms; i++) {
        aproxMonte += result[i];
        printf("Result[%d]: %f\n", i, result[i]);
    }

    printf("Monte Carlo: %.20f\n", aproxMonte / ceil_terms);
}

int main(int argc, char** argsv) {
    int n_samples = (int)get_flag_value(argc, argsv, SAMPLES_FLAG);
    int lower_bound = (int)get_flag_value(argc, argsv, LOWER_BOUND_FLAG);
    int upper_bound = (int)get_flag_value(argc, argsv, UPPER_BOUND_FLAG);

    monte_carlo(n_samples, lower_bound, upper_bound);
}
