#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <string>

#define SAMPLES_FLAG "-s"
#define LOWER_BOUND_FLAG "-l"
#define UPPER_BOUND_FLAG "-u"

#define BLOCK_SIZE 64

int get_flag_value(int argc, char **argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

float frand(float lower, float upper) {
    return (rand() / (float)RAND_MAX) * (upper - lower) + lower;
}

__device__ float fn(float x) {
    return powf(x, 2);
}

__global__ void setupKernel(hiprandState *state) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, position, 0, &state[position]);
}

__global__ void monteCarloKernel(int n_samples, float (*fn)(float), float lower, float upper, float *d_result, float *rands, hiprandState *states) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;

    d_result[position] = powf(rands[position], 2);

    printf("Postion: %d, rand_n: %f, f(rand_n): %f\n", position, rands[position], d_result[position]);
}

float monte_carlo(int n_samples, float lower, float upper) {
    int grid_size = (int)(ceil((float)n_samples / BLOCK_SIZE));
    n_samples = grid_size * BLOCK_SIZE;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size);

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));

    size_t result_size = sizeof(float) * n_samples;
    float *result = (float *)malloc(result_size);

    float *rands = (float *)malloc(result_size);

    for (int i = 0; i < n_samples; i++) {
        rands[i] = frand(lower, upper);
    }

    float *d_result, *d_rands;
    hipMalloc(&d_result, result_size);
    hipMalloc(&d_rands, result_size);

    hipMemcpy(rands, d_rands, result_size, hipMemcpyHostToDevice);

    setupKernel<<<blocksPerGrid, threadsPerBlock>>>(d_state);
    monteCarloKernel<<<blocksPerGrid, threadsPerBlock>>>(n_samples, &fn, lower, upper, d_result, d_rands, d_state);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    float integral_aprx = 0;

    for (int i = 0; i < n_samples; i++) {
        integral_aprx += result[i];
        printf("result[%d]: %f\n", i, result[i]);
    }

    printf("Blocks: %d, Threads: %d\n", grid_size, n_samples);

    return integral_aprx / (float)n_samples;
}

int main(int argc, char **argsv) {
    int n_samples = (int)get_flag_value(argc, argsv, SAMPLES_FLAG);
    float lower_bound = get_flag_value(argc, argsv, LOWER_BOUND_FLAG);
    float upper_bound = get_flag_value(argc, argsv, UPPER_BOUND_FLAG);

    srand(time(NULL));

    printf("Monte Carlo: %f\n", monte_carlo(n_samples, lower_bound, upper_bound));
}