#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <string>

#define SAMPLES_FLAG "-s"
#define LOWER_BOUND_FLAG "-l"
#define UPPER_BOUND_FLAG "-u"

#define BLOCK_SIZE 64

int get_flag_value(int argc, char **argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

__device__ half frand(float lower, float upper, int index, hiprandState *states) {
    return __float2half(hiprand_uniform(&(states[index])) * (upper - lower) + lower);
}

__device__ float fn(float x) {
    return powf(x, 2);
}

__global__ void setupKernel(hiprandState *state) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, position, 0, &state[position]);
}

__global__ void monteCarloKernel(int n_samples, float (*fn)(float), float lower, float upper, half *d_result, hiprandState *states) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;

    half rand_n = frand(lower, upper, position, states);

    d_result[position] = __float2half(powf(rand_n, 2));

    printf("Postion: %d, rand_n: %f, f(rand_n): %f\n", position, __half2float(rand_n), __half2float(d_result[position]));
}

half monte_carlo(int n_samples, float lower, float upper) {
    int grid_size = (int)(ceil((float)n_samples / BLOCK_SIZE));
    n_samples = grid_size * BLOCK_SIZE;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size);

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));

    size_t result_size = sizeof(half) * n_samples;
    half *result = (half *)malloc(result_size);
    half *d_result;

    hipMalloc(&d_result, result_size);

    setupKernel<<<blocksPerGrid, threadsPerBlock>>>(d_state);
    monteCarloKernel<<<blocksPerGrid, threadsPerBlock>>>(
        n_samples, &fn, lower, upper, d_result, d_state);

    hipDeviceSynchronize();

    hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    float integral_aprx = 0;

    for (int i = 0; i < n_samples; i++) {
        integral_aprx += __half2float(result[i]);
        printf("result[%d]: %f\n", i, __half2float(result[i]));
    }

    printf("Blocks: %d, Threads: %d\n", grid_size, n_samples);

    return __float2half(integral_aprx / n_samples);
}

int main(int argc, char **argsv) {
    int n_samples = (int)get_flag_value(argc, argsv, SAMPLES_FLAG);
    float lower_bound = get_flag_value(argc, argsv, LOWER_BOUND_FLAG);
    float upper_bound = get_flag_value(argc, argsv, UPPER_BOUND_FLAG);

    printf("Monte Carlo: %f\n", __half2float(monte_carlo(n_samples, lower_bound, upper_bound)));
}