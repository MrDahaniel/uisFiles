#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <string>

#define SAMPLES_FLAG "-s"
#define LOWER_BOUND_FLAG "-l"
#define UPPER_BOUND_FLAG "-u"

#define BLOCK_SIZE 64

int get_flag_value(int argc, char **argsv, std::string flag) {
    for (int i = 0; i < argc; i++) {
        if (flag.compare(argsv[i]) == 0) {
            return std::stoi(argsv[i + 1]);
        }
    }

    return -1;
}

__device__ float frand(float lower, float upper, int index, hiprandState *states) {
    return hiprand_uniform(&(states[index])) * (upper - lower) + lower;
}

__device__ float fn(float x) {
    return powf(x, 2);
}

__global__ void aproxMonteCarloKernel(int n_samples, float *d_result) {
    float integral_aprx = 0;
    for (int i = 0; i < n_samples; i++) {
        if (isnan(d_result[i])) {
            printf("Estalló!!!... %d %f \n", i, d_result[i]);
        } else {
            integral_aprx += d_result[i];
        }
    }
    __syncthreads();
    printf("Monte Carlo: %f", integral_aprx / (float)n_samples);
}

__global__ void setupKernel(hiprandState *state) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, position, 0, &state[position]);
}

__global__ void monteCarloKernel(int n_samples, float (*fn)(float), float lower, float upper, float *d_result, hiprandState *states) {
    int position = threadIdx.x + blockDim.x * blockIdx.x;

    float rand_n = hiprand_uniform(&(states[position])) * (upper - lower) + lower;

    d_result[position] = powf(rand_n, 2);

    if (isnan(d_result[position])) {
        // printf("Estalló!!!... %d %f \n", i, d_result[i]);
        printf("Postion: %d, rand_n: %f, f(rand_n): %f\n", position, rand_n, d_result[position]);
    }
}

void monte_carlo(int n_samples, float lower, float upper) {
    int grid_size = (int)(ceil((float)n_samples / BLOCK_SIZE));
    n_samples = grid_size * BLOCK_SIZE;

    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid(grid_size);

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));

    size_t result_size = sizeof(float) * n_samples;
    float *result = (float *)malloc(result_size);
    float *d_result;

    hipMalloc(&d_result, result_size);

    setupKernel<<<blocksPerGrid, threadsPerBlock>>>(d_state);

    hipDeviceSynchronize();

    monteCarloKernel<<<blocksPerGrid, threadsPerBlock>>>(n_samples, &fn, lower, upper, d_result, d_state);

    hipDeviceSynchronize();

    aproxMonteCarloKernel<<<1, 1>>>(n_samples, d_result);

    hipDeviceSynchronize();

    // hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    // float integral_aprx = 0;

    // for (int i = 0; i < n_samples; i++) {
    //     integral_aprx += result[i];
    //     printf("result[%d]: %f\n", i, result[i]);
    // }

    // printf("Blocks: %d, Threads: %d\n", grid_size, n_samples);

    // return integral_aprx / (float)n_samples;
}

int main(int argc, char **argsv) {
    int n_samples = (int)get_flag_value(argc, argsv, SAMPLES_FLAG);
    float lower_bound = get_flag_value(argc, argsv, LOWER_BOUND_FLAG);
    float upper_bound = get_flag_value(argc, argsv, UPPER_BOUND_FLAG);

    monte_carlo(n_samples, lower_bound, upper_bound);
}